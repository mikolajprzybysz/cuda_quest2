#include "hip/hip_runtime.h"
#include "generalCuda.cuh"
//#include <stdio.h>
//#include <math.h>
//#include <shrUtils.h>


__device__ inline int getutid()
{
int threadsPerBlock = blockDim.x * blockDim.y;
int tidWithinBlock = threadIdx.x + threadIdx.y * blockDim.x;
int gid = blockIdx.x + blockIdx.y * gridDim.x;
return gid * threadsPerBlock + tidWithinBlock;
}

__host__ __device__ double function(double x){

	double F= 0.0f;

	//tutaj wz�r funkcji
	F=0.1f*(x+3)*(x+2)*(x+1)*(x)*(x-3)*(x-5)-12;
	//F=x*x;
	return F;
}

__host__ __device__ double derivative(double x, double h){
	if(h==0){
		int j =0;//exit(-1); //printf("\n Parameter h can not be equal zero");
	};
	double d = 0.0f;
	d=(function(x+h)-function(x-h))/(2*h);
	return d;
}

__host__ __device__ float functionF(float x){

	float F= 0.0f;

	//tutaj wz�r funkcji
	F=0.1f*(x+3)*(x+2)*(x+1)*(x)*(x-3)*(x-5)-12;
	//F=x*x;
	return F;
}

__host__ __device__ float derivativeF(float x, float h){
	if(h==0){
		int j =0;//exit(-1); //printf("\n Parameter h can not be equal zero");
	};
	float d = 0.0f;
	d=(functionF(x+h)-functionF(x-h))/(2*h);
	return d;
}

__global__ void compKernel(float *d_minima,float end,float start,float incr,float h,int noOfThreads){
	int x1 = blockIdx.x*blockDim.x + threadIdx.x;
	int y1 = blockIdx.y* blockDim.y + threadIdx.y;
	int thIndex = getutid();//y1*gridDim.x +x1;
	float d =0.0f;
	float dnext = 0.0f;
	float dprev = 0.0f;
	float x = (float)start+thIndex*incr;
	
	d_minima[thIndex] = start-10.0f;
	if(x<end){
		//d_debug[thIndex]=x;
		d=derivativeF(x,h);
		dnext=derivativeF(x+incr,h);
		dprev=derivativeF(x-incr,h);

		if(thIndex>=noOfThreads ||thIndex<0) return;
		if(dprev<=0.0f && dnext>=0.0f){// ((d<threshold) && ( d>-threshold)) ){
			//if(dprev<=0.0f && dnext>=0.0f){
				d_minima[thIndex]=x;
		}else{
			d_minima[thIndex] = start-10.0f;
		}
	}
	return;


}

extern "C"  void callkernel(float* h_tableOfMinima, float end,float start, int noOfSamples,float incr, float h){
	
	float *d_minima_x = NULL;
	float *d_derivatives = NULL;
	float *d_debug = NULL;
	float *h_debug = NULL;
	int noOfThreads = (int)((end-start+incr)/incr) +1;

	noOfThreads = 16*16*32*32;
	 hipMalloc ( (void**) &d_minima_x,  noOfThreads * sizeof(float) );
	 //hipMalloc ( (void**) &d_debug,  noOfThreads * sizeof(float) );
	// h_debug = (float *) malloc(noOfThreads*sizeof(float));

	hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        printf( "Cuda error: %s: %s.\n", "error", 
                                  hipGetErrorString( err) );
    } 
	//cutilSafeCall( hipMalloc( (void**) &d_minima_x,  (noOfThreads * sizeof(float)) ));

	dim3 dimBlock(16,16);
	uint sqrtResult=(uint)sqrtl((long)noOfThreads);
	dim3 dimGrid(32,32);
	 err = hipGetLastError();
    if( hipSuccess != err) 
    {
        printf( "Cuda error: %s: %s.\n", "error", 
                                  hipGetErrorString( err) );
    }  

	uint x = dimGrid.x;

	compKernel<<<dimGrid,dimBlock>>>(d_minima_x,end,start,(end-start)/((float)noOfThreads+1.0f),h,noOfThreads/*,d_debug*/);
	//compKernel<<<dimGrid,dimBlock>>>(d_minima_x,,-3.2f,9.5f/(float)noOfThreads,0.001f,noOfThreads,d_debug);
	cutilSafeCall( hipDeviceSynchronize());
	 err = hipGetLastError();
    if( hipSuccess != err) 
    {
        printf( "Cuda error: %s: %s.\n", "error", 
                                  hipGetErrorString( err) );
    }  

	cutilSafeCall(hipMemcpy(h_tableOfMinima,d_minima_x,noOfThreads*sizeof(float), hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(h_debug,d_debug,noOfThreads*sizeof(float), hipMemcpyDeviceToHost));

	/*for(int i =0;i<noOfThreads;i++){
		 if(h_debug[i]!=0.0f)printf ("\n i = %d  val = %f ",i,h_debug[i]);
	}*/

	hipFree(d_minima_x);
}